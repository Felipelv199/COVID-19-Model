#include "hip/hip_runtime.h"
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"
#include "structures.h"

#define THREADS_N 100
#define BLOCKS_N 1

using namespace std;

__global__ void setup_kernel (hiprandState* state, unsigned long seed)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    hiprand_init ( seed, idx, 0, &state[idx] );
} 

__global__ void inicializacion_GPU(Agent* A, hiprandState* globalState, int PQ) 
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    hiprandState localState = globalState[idx];
    Agent newAgent;
    newAgent.X = (0 + hiprand_uniform(&localState) * (PQ - 0));
    newAgent.Y = (0 + hiprand_uniform(&localState) * (PQ - 0));
    newAgent.Pcon = (2 + hiprand_uniform(&localState) * (3 - 2)) / 100.0;
    newAgent.Pext = (2 + hiprand_uniform(&localState) * (3 - 2)) / 100.0;
    newAgent.Pfat = (7 + hiprand_uniform(&localState) * (70 - 7)) / 1000.0;
    newAgent.Pmov = (3 + hiprand_uniform(&localState) * (5 - 3)) / 10.0;
    newAgent.Psmo = (7 + hiprand_uniform(&localState) * (9 - 7)) / 10.0;
    newAgent.Tinc = 5 + hiprand_uniform(&localState) * (6 - 5);
    A[idx] = newAgent;
    globalState[idx] = localState;
}

__global__ void contagio_GPU(Agent* A, float r, int n, int *nuevos, float *random)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    nuevos[idx] = 0;
    Agent ai = A[idx];
    int sd = ai.S;

    if(sd != 0){
        return;
    }

    int beta = 0;
    int sigma = 0;

    for(int i=0; i<n; i++)
    {
        if(i != idx)
        {
            Agent aj = A[i];
            
            if (aj.S == 1)
            {
                beta = 1;
            }
            else
            {
                beta = 0;
            }
            
            float x = aj.X - ai.X;
            float y = aj.Y - ai.Y;
            double distance = sqrt((float)((x * x) + (y * y)));
            if (distance <= r)
            {
                //printf("aiX: %f aiY: %f ajX: %f ajY: %f distance: %f r:%f\n",ai.X, ai.Y,aj.X, aj.Y, distance, r);

                sigma += beta;
            }
        }
    }

    int alfa = 0;

    if (sigma >= 1)
    {
        alfa = 1;
    }
    float Pcond = ai.Pcon;
    //printf("random: %f Pcond: %f\n", random, Pcond);
    if (random[idx] <= Pcond)
    {
        ai.S = alfa;
        A[idx] = ai;
        if(alfa == 1)
        {
            //printf("%d entro\n", idx);
            nuevos[idx]++;
        }
        
        
        
    }
}

__global__ void movilidad_GPU(Agent* A, float pq, float lMax,
float *random1, float *random2,float *random3,float *random4,float *random5,float *random6)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    Agent ai = A[idx];
    int sd = ai.S;

    if (sd == -2)
    {
        return;
    }

    float pSmod = ai.Psmo;
    int delta = 0;

    if (random1[idx] <= pSmod)
    {
        delta = 1;
    }

    float p = pq;
    float q = pq;
    float xd = ai.X;
    float yd = ai.Y;

    float X_2 = p*random2[idx]*(1-delta);
    float X = xd + (((2*random3[idx]-1)*lMax)*delta) + X_2;
    
    float Y_2 = q*random4[idx]*(1-delta);
    float Y =  yd + (((2*random5[idx]-1)*lMax)*delta) + Y_2;
    
    //printf("X: %f Y: %f\n", X, Y);

    int gamma = 0;
    float pMovd = ai.Pmov;

    if (random6[idx] <= pMovd)
    {
        gamma = 1;
    }

    float xd1 = X;
    float yd1 = Y;

    if (xd1 > pq)
    {
        xd1 = pq - 1;
    }
    else if (xd1 < 0)
    {
        xd1 = 0;
    }

    if (yd1 > pq)
    {
        yd1 = pq - 1;
    }
    else if (yd1 < 0)
    {
        yd1 = 0;
    }

    if (gamma != 0)
    {
        ai.X = xd1;
        ai.Y = yd1;
    }
    else
    {
        ai.X = xd;
        ai.Y = yd;
    }

    A[idx] = ai;
}

__global__ void contagioExterno_GPU(Agent* A, int* R, float* random)
{
    
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
   
    R[idx] = 0;
    Agent ai = A[idx];
    int sd = ai.S;

    if(sd == 0)
    {

        int sd1 = sd;
        float pExtd = ai.Pext;
        
        //printf("id: %d %f random %fPext\n", idx, random, pExtd);

        if( random[idx] <= pExtd)
        {

            sd1 = 1;
            
        }
        
        ai.S = sd1;
        A[idx] = ai;
        if(A[idx].S == 1)
        {
            //printf("%d entro\n", idx);
            R[idx]++;
        }
    }

    
}

__global__ void tiempoIncSinCurRec_GPU(Agent* A, int *R )
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    R[idx] = 0;
    Agent ai = A[idx];
    int sd = ai.S;

    if(sd == 2 || sd == -2)
    {
        return;
    }

    int trecd1 = ai.Trec;
    

    if(sd == -1)
    {
        trecd1 -= 1;
        if (trecd1 == 0)
        {
            ai.S = 2;
            R[idx]++;
            
        }
        ai.Trec = trecd1;
        A[idx] = ai;
    }
    else if(sd == 1)
    {
        int tincd = ai.Tinc;

        tincd -= 1;
        

        int sd1 = -1;

        if (tincd > 0)
        {
            sd1 = sd;
        }
        
        
        ai.S = sd1;
        ai.Tinc = tincd;
        A[idx] = ai;
    }

}

__global__ void casosFatales_GPU(Agent* A, int* R, float *random)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    R[idx] = 0;
    Agent ai = A[idx];
    int sd = ai.S;

    if (sd == 2 || sd == -2)
    {
        return;
    }

    int rho = 0;

    if (sd < 0)
    {
        rho = 1;
    }

    int sd1 = sd;
   
    float pFatd = ai.Pfat;

    if (random[idx] <= pFatd)
    {
        if (rho > 0)
        {
            sd1 = -2;
            R[idx]++;
        }
    }

    ai.S = sd1;
    A[idx] = ai;
}

__host__ void check_CUDA_error(const char* msj) {
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("Error: %d %s (%s)\n", error, hipGetErrorString(error), msj);
	}
}

__host__ void printAgent(Agent ai)
{
    printf("X: %f, Y: %f, S: %d, Pcon: %f, Pext: %f, Pfat: %f, Pmov: %f, Psmo: %f, Tinc: %d, Trec: %d\n", ai.X, ai.Y, ai.S, ai.Pcon, ai.Pext, ai.Pfat, ai.Pmov, ai.Psmo, ai.Tinc, ai.Trec);
}

__host__ void inicializacion(int n, int pq, Agent *host_agents, hipEvent_t s, hipEvent_t e, float *et){
    Agent* dev_agents;
    hiprandState* dev_states;

    hipMalloc((void**)&dev_states, n*sizeof(hiprandState));
    check_CUDA_error("Error en hipMalloc dev_states");
    hipMalloc((void**)&dev_agents, n*sizeof(Agent));
    check_CUDA_error("Error en hipMalloc dev_agents");

    dim3 block(THREADS_N);
    dim3 grid(BLOCKS_N);
    
    hipEventRecord(s, 0);
    setup_kernel<<<grid,block>>>(dev_states, time(NULL));
	hipEventRecord(e, 0);
    check_CUDA_error("Error en kernel setup_kernel");
    hipDeviceSynchronize();
    float currElapsedTime;    
    hipEventElapsedTime(&currElapsedTime, s, e);
    *et+=currElapsedTime;
	hipEventDestroy(s);
	hipEventDestroy(e);
    
	hipEventCreate(&s);
    hipEventCreate(&e);
    hipEventRecord(s, 0);
    inicializacion_GPU<<<grid, block>>>(dev_agents, dev_states, pq);
	hipEventRecord(e, 0);
    check_CUDA_error("Error en kernel dev_agents");
    hipDeviceSynchronize();
    hipEventElapsedTime(&currElapsedTime, s, e);
    *et+=currElapsedTime;
	hipEventDestroy(s);
	hipEventDestroy(e);

    hipMemcpy(host_agents, dev_agents, n*sizeof(Agent), hipMemcpyDeviceToHost);
    check_CUDA_error("Error en hipMemcpy dev_agents-->host_agents");

    hipFree(dev_agents);
    hipFree(dev_states);
}

__host__ float rangeRandom()
{
    return (rand() % 1001 / 1000.0) ;
}

__host__ ResultsDays newDay(int movements, Agent *host_agents, Simulacion *host_simulacion, hipEvent_t s, hipEvent_t e, float *et)
{

    ResultsDays results;

    Agent* dev_agents;
    int * devcXDia;
    int * devrecupXDia;
    int * devfatXDia;

    float* devrandContagios;

    float* devRandMov1;
    float* devRandMov2;
    float* devRandMov3;
    float* devRandMov4;
    float* devRandMov5;
    float* devRandMov6;
    
    float* devrandContagiosExt;
    float* devrandFat;

    int n = host_simulacion->N;

    float r = host_simulacion->R;
    float PQ = host_simulacion->PQ;
    float lmax = host_simulacion->lmax;

    int *cXdia;
    cXdia = (int*)malloc(n*sizeof(int));
    int *recupXdia;
    recupXdia = (int*)malloc(n*sizeof(int));
    int *fatXdia;
    fatXdia = (int*)malloc(n*sizeof(int));

    float* randomContagios;
    randomContagios = (float*)malloc(n*sizeof(float));

    float* randomMov1;
    randomMov1 = (float*)malloc(n*sizeof(float));
    float* randomMov2;
    randomMov2 = (float*)malloc(n*sizeof(float));
    float* randomMov3;
    randomMov3 = (float*)malloc(n*sizeof(float));
    float* randomMov4;
    randomMov4 = (float*)malloc(n*sizeof(float));
    float* randomMov5;
    randomMov5 = (float*)malloc(n*sizeof(float));
    float* randomMov6;
    randomMov6 = (float*)malloc(n*sizeof(float));

    float* randomContagiosExt;
    randomContagiosExt = (float*)malloc(n*sizeof(float));
    float* randomFat;
    randomFat = (float*)malloc(n*sizeof(float));

    dim3 block(THREADS_N);
    dim3 grid(BLOCKS_N);

    hipMalloc((void**)&dev_agents, n*sizeof(Agent));
    check_CUDA_error("Error en hipMalloc dev_agents");
    hipMalloc((void**)&devcXDia, n*sizeof(int));
    check_CUDA_error("Error en hipMalloc devDia");
    hipMalloc((void**)&devrecupXDia, n*sizeof(int));
    check_CUDA_error("Error en hipMalloc devrecupXDia");
    hipMalloc((void**)&devfatXDia, n*sizeof(int));
    check_CUDA_error("Error en hipMalloc devfatXDia");
    
    hipMalloc((void**)&devrandContagios, n*sizeof(float));
    check_CUDA_error("Error en hipMalloc devrandContagios");

    hipMalloc((void**)&devRandMov1, n*sizeof(float));
    check_CUDA_error("Error en hipMalloc devRandMov1");
    hipMalloc((void**)&devRandMov2, n*sizeof(float));
    check_CUDA_error("Error en hipMalloc devRandMov2");
    hipMalloc((void**)&devRandMov3, n*sizeof(float));
    check_CUDA_error("Error en hipMalloc devRandMov3");
    hipMalloc((void**)&devRandMov4, n*sizeof(float));
    check_CUDA_error("Error en hipMalloc devRandMov4");
    hipMalloc((void**)&devRandMov5, n*sizeof(float));
    check_CUDA_error("Error en hipMalloc devRandMov5");
    hipMalloc((void**)&devRandMov6, n*sizeof(float));
    check_CUDA_error("Error en hipMalloc devRandMov6");

    hipMalloc((void**)&devrandContagiosExt, n*sizeof(float));
    check_CUDA_error("Error en hipMalloc devrandContagios");
    hipMalloc((void**)&devrandFat, n*sizeof(float));
    check_CUDA_error("Error en hipMalloc devrandContagios"); 

    for (int j = 0; j < movements; j++)
    {
        hipMemcpy(dev_agents, host_agents, n*sizeof(Agent), hipMemcpyHostToDevice);
        check_CUDA_error("Error en hipMalloc host_agents-->dev_agents");
        hipMemcpy(devcXDia, cXdia, n*sizeof(float), hipMemcpyHostToDevice);
        check_CUDA_error("Error en hipMalloc cXdia-->devcXDia");

        for(int i = 0; i < n; i++)
        {
            randomContagios[i]  = rangeRandom();
            randomMov1[i] = rangeRandom();
            randomMov2[i] = rangeRandom();
            randomMov3[i] = rangeRandom();
            randomMov4[i] = rangeRandom();
            randomMov5[i] = rangeRandom();
            randomMov6[i] = rangeRandom();
        }

        hipMemcpy(devrandContagios, randomContagios, n*sizeof(float), hipMemcpyHostToDevice);
        check_CUDA_error("Error en hipMalloc randomContagios-->devrandContagios");

        hipMemcpy(devRandMov1, randomMov1, n*sizeof(float), hipMemcpyHostToDevice);
        check_CUDA_error("Error en hipMalloc randomContagios-->devrandContagios");
        hipMemcpy(devRandMov2, randomMov2, n*sizeof(float), hipMemcpyHostToDevice);
        check_CUDA_error("Error en hipMalloc randomContagios-->devrandContagios");
        hipMemcpy(devRandMov3, randomMov3, n*sizeof(float), hipMemcpyHostToDevice);
        check_CUDA_error("Error en hipMalloc randomContagios-->devrandContagios");
        hipMemcpy(devRandMov4, randomMov4, n*sizeof(float), hipMemcpyHostToDevice);
        check_CUDA_error("Error en hipMalloc randomContagios-->devrandContagios");
        hipMemcpy(devRandMov5, randomMov5, n*sizeof(float), hipMemcpyHostToDevice);
        check_CUDA_error("Error en hipMalloc randomContagios-->devrandContagios");
        hipMemcpy(devRandMov6, randomMov6, n*sizeof(float), hipMemcpyHostToDevice);
        check_CUDA_error("Error en hipMalloc randomContagios-->devrandContagios");
        
        hipEventCreate(&s);
        hipEventCreate(&e);
        hipEventRecord(s, 0);
        contagio_GPU<<<grid, block>>>(dev_agents, r, n, devcXDia, devrandContagios);
	    hipEventRecord(e, 0);
        check_CUDA_error("Error en kernel contagio_GPU");
        hipDeviceSynchronize();
        float currElapsedTime;  
        hipEventElapsedTime(&currElapsedTime, s, e);
        *et+=currElapsedTime;
        hipEventDestroy(s);
        hipEventDestroy(e);

        hipEventCreate(&s);
        hipEventCreate(&e);
        hipEventRecord(s, 0);
        movilidad_GPU<<<grid, block>>>(dev_agents, PQ, lmax, devRandMov1, devRandMov2, devRandMov3, devRandMov4, devRandMov5, devRandMov6);
	    hipEventRecord(e, 0);
        check_CUDA_error("Error en kernel movilidad_GPU");
        hipDeviceSynchronize();  
        hipEventElapsedTime(&currElapsedTime, s, e);
        *et+=currElapsedTime;
        hipEventDestroy(s);
        hipEventDestroy(e);

        hipMemcpy(cXdia, devcXDia, n*sizeof(int), hipMemcpyDeviceToHost);
        check_CUDA_error("Error en hipMalloc devcXDia-->cXDia");
        for(int j =0; j < n; j++)
        {
            results.c+= cXdia[j];
            //printf("casos:%d\n", cXdia[j]);
        }
        hipMemcpy(host_agents, dev_agents, n*sizeof(Agent), hipMemcpyDeviceToHost);
        check_CUDA_error("Error en hipMalloc devAgents-->host_agents");

    }
    hipMemcpy(dev_agents, host_agents, n*sizeof(Agent), hipMemcpyHostToDevice);
    check_CUDA_error("Error en hipMalloc host_agents-->dev_agents");
    hipMemcpy(devcXDia, cXdia, n*sizeof(int), hipMemcpyHostToDevice);
    check_CUDA_error("Error en hipMalloc cXdia-->devcXDia");

    hipMemcpy(devrecupXDia, recupXdia, n*sizeof(int), hipMemcpyHostToDevice);
    check_CUDA_error("Error en hipMalloc recupXdia-->devrecupXDia");
    hipMemcpy(devfatXDia, fatXdia, n*sizeof(int), hipMemcpyHostToDevice);
    check_CUDA_error("Error en hipMalloc fatXdia-->devfatXDia");

    for(int i = 0; i < n; i++)
    {
        randomContagiosExt[i]  = rangeRandom();
        randomFat[i]  = rangeRandom();
    }    
    
    hipMemcpy(devrandContagiosExt, randomContagiosExt, n*sizeof(float), hipMemcpyHostToDevice);
    check_CUDA_error("Error en hipMalloc randomContagios-->devrandContagios");
    hipMemcpy(devrandFat, randomFat, n*sizeof(float), hipMemcpyHostToDevice);
    check_CUDA_error("Error en hipMalloc randomContagios-->devrandContagios");

    hipEventCreate(&s);
    hipEventCreate(&e);
    hipEventRecord(s, 0);
    contagioExterno_GPU<<<grid,block>>>(dev_agents, devcXDia, devrandContagiosExt);
    hipEventRecord(e, 0);
    check_CUDA_error("Error en kernel contagioExterno_GPU");
    hipDeviceSynchronize();
    float currElapsedTime; 
    hipEventElapsedTime(&currElapsedTime, s, e);
    *et+=currElapsedTime;
    hipEventDestroy(s);
    hipEventDestroy(e);
    hipMemcpy(cXdia, devcXDia, n*sizeof(int), hipMemcpyDeviceToHost);
    check_CUDA_error("Error en hipMalloc devcXDia-->cXDia");

    hipMemcpy(host_agents, dev_agents, n*sizeof(Agent), hipMemcpyDeviceToHost);
    check_CUDA_error("Error en hipMalloc devAgents-->host_agents");

    hipEventCreate(&s);
    hipEventCreate(&e);
    hipEventRecord(s, 0);
    tiempoIncSinCurRec_GPU<<<grid,block>>>(dev_agents, devrecupXDia);
    hipEventRecord(e, 0);
    check_CUDA_error("Error en kernel tiempoIncSinCurRec_GPU");
    hipDeviceSynchronize(); 
    hipEventElapsedTime(&currElapsedTime, s, e);
    *et+=currElapsedTime;
    hipEventDestroy(s);
    hipEventDestroy(e);

    hipEventCreate(&s);
    hipEventCreate(&e);
    hipEventRecord(s, 0);
    casosFatales_GPU<<<grid,block>>>(dev_agents, devfatXDia, devrandFat);
    hipEventRecord(e, 0);
    check_CUDA_error("Error en kernel casosFatales_GPU");
    hipDeviceSynchronize();
    hipEventElapsedTime(&currElapsedTime, s, e);
    *et+=currElapsedTime;
    hipEventDestroy(s);
    hipEventDestroy(e);

    hipMemcpy(cXdia, devcXDia, n*sizeof(int), hipMemcpyDeviceToHost);
    check_CUDA_error("Error en hipMalloc devcXDia-->cXDia");
    hipMemcpy(recupXdia, devrecupXDia, n*sizeof(int), hipMemcpyDeviceToHost);
    check_CUDA_error("Error en hipMalloc devcXDia-->cXDia");
    hipMemcpy(fatXdia, devfatXDia, n*sizeof(int), hipMemcpyDeviceToHost);
    check_CUDA_error("Error en hipMalloc devcXDia-->cXDia");
    hipMemcpy(host_agents, dev_agents, n*sizeof(Agent), hipMemcpyDeviceToHost);
    check_CUDA_error("Error en hipMalloc devAgents-->host_agents");
    
    for(int j =0; j < n; j++)
    {
        results.c+= cXdia[j];
        results.cRecup += recupXdia[j];
        results.cFat += fatXdia[j];
    //printf("casos:%d\n", cXdia[j]);
    }
    

    hipFree(dev_agents);

    hipFree(devrandContagios);
    free(randomContagios);

    hipFree(devRandMov1);
    free(randomMov1);
    hipFree(devRandMov2);
    free(randomMov2);
    hipFree(devRandMov3);
    free(randomMov3);
    hipFree(devRandMov4);
    free(randomMov4);
    hipFree(devRandMov5);
    free(randomMov5);
    hipFree(devRandMov6);
    free(randomMov6);

    hipFree(devrandContagiosExt);
    free(randomContagiosExt);
    hipFree(devrandFat);
    free(randomFat);

    hipFree(devcXDia);
    free(cXdia);

    hipFree(devrecupXDia);
    free(recupXdia);

    hipFree(devfatXDia);
    free(fatXdia);

    return results;
}

int main(){

    FILE * file;
    file = fopen("results.txt", "w");
    if(file == NULL)
    {
        /* File not created hence exit */
        printf("Unable to create results file.\n");
        exit(EXIT_FAILURE);
    }
    fprintf (file, "\n----------------------------------------------\nResults\n---------------------------------------------- \n");

    const int N = THREADS_N * BLOCKS_N;
    const int DAYS = 31;
    Simulacion simulacion;
    simulacion.N = N;
    simulacion.dmax = DAYS;
    int mM = simulacion.Mmax;
    Agent* agents;    
	hipEvent_t start;
    hipEvent_t end;
    float elapsedTime=0;

    agents = (Agent*)malloc(N*sizeof(Agent));
	hipEventCreate(&start);
    hipEventCreate(&end);

    inicializacion(N, simulacion.PQ, agents, start, end, &elapsedTime);
    
    ResultsDays *results;
    results = (ResultsDays*)malloc(DAYS*sizeof(ResultsDays));

    for(int i=1; i<=DAYS; i++)
    {
        

        results[i-1] = newDay(mM, agents, &simulacion, start, end, &elapsedTime);
        
        simulacion.results.cAcum += results[i-1].c;
        simulacion.results.cAcumAgRecup += results[i-1].cRecup;
        simulacion.results.cFatAcum += results[i-1].cFat;
        if (simulacion.results.cAcum == results[i-1].c && simulacion.results.cAcum > 0)
        {
            simulacion.results.cZero = i;
        }
        //printf("%d es aqui, %d,%d\n", i, simulacion.results.cAcumAgRecup, simulacion.results.cRecupXDia);
        if (simulacion.results.cAcumAgRecup == results[i-1].cRecup && simulacion.results.cAcumAgRecup > 0)
        {
            
            simulacion.results.recupPrim = i;
        }

        if (simulacion.results.cFatAcum == results[i-1].cFat && simulacion.results.cFatAcum > 0)
        {
            //printf("%d es aqui tambien\n", i);
            simulacion.results.cFatPrim = i;
        }

        
        printf("Dia %d\n", i);
        fprintf (file, "Dia %d\n", i);
        
        printf("    Numero de nuevos casos positivos por dia: %d\n", results[i-1].c);
        fprintf (file, "    Numero de nuevos casos positivos por dia: %d\n", results[i-1].c);

        printf("    Numero de casos recuperados por dia: %d\n", results[i-1].cRecup);
        fprintf (file, "    Numero de casos recuperados por dia: %d\n", results[i-1].cRecup);

        printf("    Numero de casos fatales por dia: %d\n", results[i-1].cFat);
        fprintf (file, "    Numero de casos fatales por dia: %d\n", results[i-1].cFat);
        
        printf("------------------------\n");
        fprintf (file, "------------------------\n");
        
    }

    printf("Resultados Finales de %d Dias\n", DAYS);
    fprintf (file, "Resultados Finales de %d Dias\n", DAYS);

    printf("    Numero agentes: %d\n", N);
    fprintf (file, "    Numero agentes: %d\n", N);

    printf("    Numero de casos acumulados de agentes contagiados: %d\n", simulacion.results.cAcum);
    fprintf (file, "    Numero de casos acumulados de agentes contagiados: %d\n", simulacion.results.cAcum);

    printf("    Numero de casos acumulados de agentes recuperados: %d\n", simulacion.results.cAcumAgRecup);
    fprintf (file, "    Numero de casos acumulados de agentes recuperados: %d\n", simulacion.results.cAcumAgRecup);

    printf("    Numero de casos fatales acumulados: %d\n", simulacion.results.cFatAcum);
    fprintf (file, "    Numero de casos fatales acumulados: %d\n", simulacion.results.cFatAcum);


    int acum50 = simulacion.results.cAcum/2;
    int acum100 = simulacion.results.cAcum;

    int recup50 = simulacion.results.cAcumAgRecup / 2;
    int recup100 = simulacion.results.cAcumAgRecup;

    int fat50 = simulacion.results.cFatAcum / 2;
    int fat100 = simulacion.results.cFatAcum;

    int c = 0;
    int rec = 0;
    int f = 0;
    for(int i = 0; i < DAYS; i++)
    {
        c += results[i].c;
        if(c >= acum50 && simulacion.results.c50per == 0 && simulacion.results.cAcum > 0)
        {
            simulacion.results.c50per = i+1;
        }

        if(c >= acum100 && simulacion.results.c100per == 0 && simulacion.results.cAcum > 0)
        {
            simulacion.results.c100per = i+1;
        }
        rec+= results[i].cRecup;
        if(rec >= recup50 && simulacion.results.recup50per == 0 && simulacion.results.cAcumAgRecup > 0)
        {
            simulacion.results.recup50per = i+1;
        }

        if(rec >= recup100 && simulacion.results.recup100per == 0 && simulacion.results.cAcumAgRecup > 0)
        {
            simulacion.results.recup100per =i+1;
        }
        f += results[i].cFat;
        if(f >= fat50 && simulacion.results.cFat50per == 0 && simulacion.results.cFatAcum > 0)
        {
            simulacion.results.cFat50per = i+1;
        }

        if(f >= fat100 && simulacion.results.cFat100per == 0 && simulacion.results.cFatAcum > 0)
        {
            simulacion.results.cFat100per = i+1;
        }
    }




    if(simulacion.results.cZero == 0)
    {
        printf("    Dia en que se contagio el primer agente: N/A");
        fprintf (file,"    Dia en que se contagio el primer agente: N/A" );
    }
    else{
        printf("    Dia en que se contagio el primer agente: %d\n", simulacion.results.cZero);
        fprintf (file, "    Dia en que se contagio el primer agente: %d\n", simulacion.results.cZero);
    }

    if(simulacion.results.c50per == 0)
    {
        printf("    Dia en que se contagio el 50%% de los agentes contagiados: N/A\n");
        fprintf (file, "    Dia en que se contagio el 50%% de los agentes contagiados: N/A\n");

    }
    else{
        printf("    Dia en que se contagio el 50%% de los agentes contagiados: %d\n", simulacion.results.c50per);
        fprintf (file,"    Dia en que se contagio el 50%% de los agentes contagiados: %d\n", simulacion.results.c50per );

    }

    if(simulacion.results.c50per == 0)
    {
        printf("    Dia en que se contagio el 100%% de los agentes contagiados: N/A\n");
        fprintf (file, "    Dia en que se contagio el 100%% de los agentes contagiados: N/A\n");

    }
    else{
        printf("    Dia en que se contagio el 100%% de los agentes contagiados: %d\n", simulacion.results.c100per);
        fprintf (file, "    Dia en que se contagio el 100%% de los agentes contagiados: %d\n", simulacion.results.c100per);

    }

    if(simulacion.results.recupPrim == 0)
    {
        printf("    Dia en que se recupero el primer agente: N/A\n");
        fprintf (file,"    Dia en que se recupero el primer agente: N/A\n" );
    }
    else{
        printf("    Dia en que se recupero el primer agente: %d\n", simulacion.results.recupPrim);
        fprintf (file, "    Dia en que se recupero el primer agente: %d\n", simulacion.results.recupPrim);
    }
    
    if(simulacion.results.recup50per == 0)
    {
        printf("    Dia en que se recupero el 50%% de los agentes recuperados: N/A\n");
        fprintf (file,"    Dia en que se recupero el 50%% de los agentes recuperados: N/A\n" );
    }
    else{
        printf("    Dia en que se recupero el 50%% de los agentes recuperados: %d\n", simulacion.results.recup50per);
        fprintf (file,"    Dia en que se recupero el 50%% de los agentes recuperados: %d\n", simulacion.results.recup50per );
    }

    if(simulacion.results.recup100per == 0)
    {
        printf("    Dia en que se recupero el 100%% de los agentes recuperados: N/A\n");
        fprintf (file, "    Dia en que se recupero el 100%% de los agentes recuperados: N/A\n");
    }
    else{
        printf("    Dia en que se recupero el 100%% de los agentes recuperados: %d\n", simulacion.results.recup100per);
        fprintf (file, "    Dia en que se recupero el 100%% de los agentes recuperados: %d\n", simulacion.results.recup100per);
    }

    if(simulacion.results.cFatPrim == 0)
    {
        printf("    Dia en que ocurrio el primer caso fatal: N/A\n");
        fprintf (file,"    Dia en que ocurrio el primer caso fatal: N/A\n" );
    }
    else{
        printf("    Dia en que ocurrio el primer caso fatal: %d\n", simulacion.results.cFatPrim);
        fprintf (file, "    Dia en que ocurrio el primer caso fatal: %d\n", simulacion.results.cFatPrim);
    }

    if(simulacion.results.cFat50per == 0)
    {
        printf("    Dia en que ocurrio el 50%% de los casos fatales: N/A\n");
        fprintf (file, "    Dia en que ocurrio el 50%% de los casos fatales: N/A\n");
    }
    else{
        printf("    Dia en que ocurrio el 50%% de los casos fatales: %d\n", simulacion.results.cFat50per);
        fprintf (file,"    Dia en que ocurrio el 50%% de los casos fatales: %d\n", simulacion.results.cFat50per );
    }
    
    if(simulacion.results.cFat100per == 0)
    {
        printf("    Dia en que ocurrio el 100%% de los casos fatales: N/A\n");
        fprintf (file, "    Dia en que ocurrio el 100%% de los casos fatales: N/A\n");
    }
    else{
        printf("    Dia en que ocurrio el 100%% de los casos fatales: %d\n", simulacion.results.cFat100per);
        fprintf (file,"    Dia en que ocurrio el 100%% de los casos fatales: %d\n", simulacion.results.cFat100per );
    }
    printf("Time GPU: %f miliseconds.\n", elapsedTime);
    fprintf (file, "Time GPU: %f miliseconds.\n", elapsedTime);
    
    printf("------------------------\n");
    fprintf (file, "------------------------\n");
    free(agents);
    free(results);
    
    return 0;
}
