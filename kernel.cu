#include "hip/hip_runtime.h"
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"

#include "kernel.h"

using namespace std;

__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    hiprand_init ( seed, idx, 0, &state[idx] );
} 

__global__ void inicializacion_GPU(Agent* A, hiprandState* globalState, int PQ) 
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    hiprandState localState = globalState[idx];
    Agent newAgent;
    newAgent.X = (int)(0 + hiprand_uniform(&localState) * (PQ - 0));
    newAgent.Y = (int)(0 + hiprand_uniform(&localState) * (PQ - 0));
    newAgent.Pcon = (2 + hiprand_uniform(&localState) * (3 - 2)) / 100.0;
    newAgent.Pext = (2 + hiprand_uniform(&localState) * (3 - 2)) / 100.0;
    newAgent.Pfat = (7 + hiprand_uniform(&localState) * (70 - 7)) / 1000.0;
    newAgent.Pmov = (3 + hiprand_uniform(&localState) * (5 - 3)) / 10.0;
    newAgent.Psmo = (7 + hiprand_uniform(&localState) * (9 - 7)) / 10.0;
    newAgent.Tinc = 5 + hiprand_uniform(&localState) * (6 - 5);
    A[idx] = newAgent;
    globalState[idx] = localState;
}

__host__ void check_CUDA_error(const char* msj) {
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("Error: %d %s (%s)\n", error, hipGetErrorString(error), msj);
	}
}

__host__ void inicializacion(int n, int pq, Agent *host_agents){
    hiprandState* dev_states;
    Agent* dev_agents;

    hipMalloc((void**)&dev_states, n*sizeof(hiprandState));
    check_CUDA_error("Error en hipMalloc dev_states");
    hipMalloc((void**)&dev_agents, n*sizeof(Agent));
    check_CUDA_error("Error en hipMalloc dev_agents");

    dim3 block(1024);
    dim3 grid(10);

    setup_kernel<<<grid,block>>>(dev_states, time(NULL));
    check_CUDA_error("Error en kernel setup_kernel");
    hipDeviceSynchronize();
    inicializacion_GPU<<<grid, block>>>(dev_agents, dev_states, pq);
    check_CUDA_error("Error en kernel dev_agents");
    hipDeviceSynchronize();

    hipMemcpy(host_agents, dev_agents, n*sizeof(Agent), hipMemcpyDeviceToHost);
    check_CUDA_error("Error en hipMemcpy dev_agents-->host_agents");
    
    for(int i=0; i<n; i++){
        Agent ai = host_agents[n];
        printf("X:%d Y:%d Pcon:%f Pext:%f Pfat:%f Pmov:%f Psmo:%f Tinc:%d\n", ai.X, ai.Y, ai.Pcon, ai.Pext, ai.Pfat, ai.Pmov, ai.Psmo, ai.Tinc);
    } 

    hipFree(dev_agents);
    hipFree(dev_states);
}

int main(){
    const int N = 10240;
    const int DAYS = 100;
    Simulacion sim;
    sim.N = N;
    sim.dmax = DAYS;
    Agent* agents;

    agents = (Agent*)malloc(N*sizeof(Agent));

    inicializacion(N, sim.PQ, agents);

    free(agents);
    return 0;
}