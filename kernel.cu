#include "hip/hip_runtime.h"
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"

#include "kernel.h"

#define THREADS_N 1024
#define BLOCKS_N 10

using namespace std;

__global__ void setup_kernel (hiprandState* state, unsigned long seed)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    hiprand_init ( seed, idx, 0, &state[idx] );
} 

__global__ void inicializacion_GPU(Agent* A, hiprandState* globalState, int PQ) 
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    hiprandState localState = globalState[idx];
    Agent newAgent;
    newAgent.X = (int)(0 + hiprand_uniform(&localState) * (PQ - 0));
    newAgent.Y = (int)(0 + hiprand_uniform(&localState) * (PQ - 0));
    newAgent.Pcon = (2 + hiprand_uniform(&localState) * (3 - 2)) / 100.0;
    newAgent.Pext = (2 + hiprand_uniform(&localState) * (3 - 2)) / 100.0;
    newAgent.Pfat = (7 + hiprand_uniform(&localState) * (70 - 7)) / 1000.0;
    newAgent.Pmov = (3 + hiprand_uniform(&localState) * (5 - 3)) / 10.0;
    newAgent.Psmo = (7 + hiprand_uniform(&localState) * (9 - 7)) / 10.0;
    newAgent.Tinc = 5 + hiprand_uniform(&localState) * (6 - 5);
    A[idx] = newAgent;
    globalState[idx] = localState;
}

__global__ void contagio_GPU(Agent* A, hiprandState* globalState, int r, int n)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    hiprandState localState = globalState[idx];
    Agent ai = A[idx];
    int sd = ai.S;

    if(sd != 0){
        return;
    }

    int beta = 0;
    int sigma = 0;

    for(int i=0; i<n; i++)
    {
        if(i != idx)
        {
            Agent aj = A[i];
            if (aj.S > 0)
            {
                beta = 1;
            }
            else
            {
                beta = 0;
            }
            
            int x = ai.X;
            int y = ai.Y;
            double distance = sqrt((float)(x * x) + (float)(y * y));

            if (distance <= r)
            {
                sigma += distance * beta;
            }
        }
    }

    int alfa = 0;

    if (sigma >= 1)
    {
        alfa = 1;
    }

    float random = (hiprand_uniform(&localState) * 100) / 100.0;
    int Pcond = ai.Pcon;

    if (random <= Pcond)
    {
        ai.S = random * alfa;
        A[idx] = ai;
    }
}

__host__ void check_CUDA_error(const char* msj) {
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("Error: %d %s (%s)\n", error, hipGetErrorString(error), msj);
	}
}

__host__ void inicializacion(int n, int pq, Agent *host_agents){
    Agent* dev_agents;
    hiprandState* dev_states;

    hipMalloc((void**)&dev_states, n*sizeof(hiprandState));
    check_CUDA_error("Error en hipMalloc dev_states");
    hipMalloc((void**)&dev_agents, n*sizeof(Agent));
    check_CUDA_error("Error en hipMalloc dev_agents");

    dim3 block(THREADS_N);
    dim3 grid(BLOCKS_N);

    setup_kernel<<<grid,block>>>(dev_states, time(NULL));
    check_CUDA_error("Error en kernel setup_kernel");
    hipDeviceSynchronize();
    inicializacion_GPU<<<grid, block>>>(dev_agents, dev_states, pq);
    check_CUDA_error("Error en kernel dev_agents");
    hipDeviceSynchronize();

    hipMemcpy(host_agents, dev_agents, n*sizeof(Agent), hipMemcpyDeviceToHost);
    check_CUDA_error("Error en hipMemcpy dev_agents-->host_agents");

    hipFree(dev_agents);
    hipFree(dev_states);
}

__host__ void contagio(Agent *host_agents, Simulacion *host_simulacion, int n){
    Agent* dev_agents;
    hiprandState* dev_states;

    hipMalloc((void**)&dev_agents, n*sizeof(Agent));
    check_CUDA_error("Error en hipMalloc dev_agents");
    hipMalloc((void**)&dev_states, n*sizeof(hiprandState));
    check_CUDA_error("Error en hipMalloc dev_states");

    hipMemcpy(dev_agents, host_agents, n*sizeof(Agent), hipMemcpyHostToDevice);
    check_CUDA_error("Error en hipMalloc host_agents-->dev_agents");

    dim3 block(THREADS_N);
    dim3 grid(BLOCKS_N);

    setup_kernel<<<grid,block>>>(dev_states, time(NULL));
    check_CUDA_error("Error en kernel setup_kernel");
    hipDeviceSynchronize();
    contagio_GPU<<<grid, block>>>(dev_agents, dev_states, host_simulacion->R, host_simulacion->N);
    check_CUDA_error("Error en kernel contagio_GPU");
    hipDeviceSynchronize();

    hipMemcpy(host_agents, dev_agents, n*sizeof(Agent), hipMemcpyDeviceToHost);
    check_CUDA_error("Error en hipMalloc host_agents-->dev_agents");

    hipFree(dev_agents);
    hipFree(dev_states);
}

__host__ void printAgent(Agent ai)
{
    printf("X: %d, Y: %d, S: %d, Pcon: %f, Pext: %f, Pfat: %f, Pmov: %f, Psmo: %f, Tinc: %d\n", ai.X, ai.Y, ai.S, ai.Pcon, ai.Pext, ai.Pfat, ai.Pmov, ai.Psmo, ai.Tinc);
}

int main(){
    const int N = 10240;
    const int DAYS = 100;
    Simulacion simulacion;
    simulacion.N = N;
    simulacion.dmax = DAYS;
    int mM = simulacion.Mmax;
    Agent* agents;

    agents = (Agent*)malloc(N*sizeof(Agent));

    inicializacion(N, simulacion.PQ, agents);

    for(int i=0; i<DAYS; i++)
    {
        for (int j = 0; j < mM; j++)
        {   
            contagio(agents, &simulacion, N);
        }
    } 

    free(agents);
    return 0;
}